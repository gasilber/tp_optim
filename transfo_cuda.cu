#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include "transfo_cuda.h"

#define DIM 256

__global__ void copyKernel(int w, int h, unsigned char *src, unsigned char *dest)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < w && j < h)
    {
        dest[j * w + i] = src[j * w + i];
    }
}

extern "C" void copyGPU(int w, int h, unsigned char *src, unsigned char *dest)
{
    // Define block and grid dimensions
    dim3 blockDim(DIM, DIM); // Adjust according to your GPU architecture
    dim3 gridDim((w + blockDim.x - 1) / blockDim.x, (h + blockDim.y - 1) / blockDim.y);

    // Allocate device memory
    unsigned char *d_src, *d_dest;
    hipMalloc((void **)&d_src, w * h * sizeof(unsigned char));
    hipMalloc((void **)&d_dest, w * h * sizeof(unsigned char));

    // Copy input data from host to device
    hipMemcpy(d_src, src, w * h * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_dest, dest, w * h * sizeof(unsigned char), hipMemcpyHostToDevice);


    // Launch the kernel
    copyKernel<<<gridDim, blockDim>>>(w, h, d_src, d_dest);

    // Copy the result back to the host
    hipMemcpy(dest, d_dest, w * h * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_src);
    hipFree(d_dest);
}

__global__ void lightKernel(int w, int h, unsigned char *img, unsigned char val)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < w && j < h)
    {
        unsigned char current = img[j * w + i];
        img[j * w + i] = ((int)current + val > 255) ? 255 : current + val;
    }
}

extern "C" void lightGPU(int w, int h, unsigned char *img, unsigned char val)
{
    dim3 blockDim(DIM, DIM); // Adjust according to your GPU architecture
    dim3 gridDim((w + blockDim.x - 1) / blockDim.x, (h + blockDim.y - 1) / blockDim.y);

    // Allocate device memory
    unsigned char *d_img;
    hipMalloc((void **)&d_img, w * h * sizeof(unsigned char));

    // Copy input data from host to device
    hipMemcpy(d_img, img, w * h * sizeof(unsigned char), hipMemcpyHostToDevice);

    lightKernel<<<gridDim, blockDim>>>(w, h, d_img, val);

    // Copy the result back to the host
    hipMemcpy(img, d_img, w * h * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_img);
}

__global__ void curveKernel(int w, int h, unsigned char *img, unsigned char *lut)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < w && j < h)
    {
        unsigned char current = img[j * w + i];
        img[j * w + i] = lut[current];
    }
}

extern "C" void curveGPU(int w, int h, unsigned char *img, unsigned char *lut)
{
    dim3 blockDim(DIM, DIM); // Adjust according to your GPU architecture
    dim3 gridDim((w + blockDim.x - 1) / blockDim.x, (h + blockDim.y - 1) / blockDim.y);

    // Allocate device memory
    unsigned char *d_img, *d_lut;
    hipMalloc((void **)&d_img, w * h * sizeof(unsigned char));
    hipMalloc((void **)&d_lut, w * h * sizeof(unsigned char));

    // Copy input data from host to device
    hipMemcpy(d_img, img, w * h * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_lut, lut, w * h * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Launch the kernel
    curveKernel<<<gridDim, blockDim>>>(w, h, d_img, d_lut);

    // Copy the result back to the host
    hipMemcpy(img, d_img, w * h * sizeof(unsigned char), hipMemcpyDeviceToHost);
    hipMemcpy(lut, d_lut, w * h * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_img);
    hipFree(d_lut);
}